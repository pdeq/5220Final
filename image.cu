#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define NUM_THREADS 256

// Serial. Parallelize and put on GPU later
int *three_kernel(int *color_array, float *mask, int num_frames, int height, int width){
    int *masked_array = (int *) malloc(num_frames * height * width * sizeof(int));
    for (int i = 0; i < num_frames; ++i){
        for (int j = 1; j < height - 1; ++j){ // Avoid edges (at least for now)
            for (int k = 1; k < width - 1; ++k){ // Avoid edges
                int index = i * (width * height) + j * (width) + k;
                float acc = 0;
                acc += (mask[4] * color_array[index]);
                acc += (mask[5] * color_array[index + 1] + mask[3] * color_array[index - 1]); // Horizontal neighbors
                acc += (mask[7] * color_array[index + width] + mask[1] * color_array[index - width]); // Vertical neighbors
                acc += (mask[8] * color_array[index + width + 1] + mask[2] * color_array[index - width + 1]); // Right diag
                acc += (mask[6] * color_array[index + width - 1] + mask[0] * color_array[index - width - 1]); // Left diag
                masked_array[index] = (int) (acc);
            }
        }
    }
    return masked_array;
}

// Serial. Parallelize and put on GPU later
void tint_color(int *color_array, int color_val, float weight, int array_len){
    for (int i = 0; i < array_len; ++i){
        color_array[i] = (int) ((color_val + weight * color_array[i]) > 255 ? 255 : (color_val + weight * color_array[i]));

    }
}

// Serial. Parallelize and put on GPU later
void shade_color(int *color_array, int color_val, float weight, int array_len){
    for (int i = 0; i < array_len; ++i){
        color_array[i] = (int) ((color_val - weight * color_array[i]) < 0 ? 0 : (color_val - weight * color_array[i]));

    }
}

__global__ void d_tint_color(int* d_color_array, int color_val, float weight, int array_len) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= array_len) return;

    d_color_array[tid] = (int) ((color_val + weight * d_color_array[tid]) > 255 ? 255 : (color_val + weight * d_color_array[tid]));
}

